#include "hip/hip_runtime.h"
// 
// Vector Reduction
//

// Includes
#include <stdio.h>
#include <cutil_inline.h>

// Input Array Variables
float* h_In = NULL;
float* d_In = NULL;

// Output Array
float* h_Out = NULL;
float* d_Out = NULL;

// Variables to change
int GlobalSize = 50000;
int BlockSize = 32;

// Functions
void Cleanup(void);
void RandomInit(float*, int);
void PrintArray(float*, int);
float CPUReduce(float*, int);
void ParseArguments(int, char**);

//timers
unsigned int timer_CPU = 0; //time to calculate partial sums in cpu
unsigned int timer_GPU = 0;
unsigned int timer_mem = 0;
unsigned int timer_total = 0;

// Device code
__global__ void VecReduce(float* g_idata, float* g_odata, int N)
{
  // shared memory size declared at kernel launch
  extern __shared__ float sdata[]; 

  unsigned int tid = threadIdx.x; 
  unsigned int globalid = blockIdx.x*blockDim.x + threadIdx.x; 

  // For thread ids greater than data space
  sdata[tid] = g_idata[globalid]; 


  // each thread loads one element from global to shared mem
  __syncthreads();

  // do reduction in shared mem
  for (unsigned int s=blockDim.x / 2; s > 0; s = s >> 1) {
     if (tid < s) { 
         sdata[tid] = sdata[tid] + sdata[tid+ s];
     }
     __syncthreads();
  }

  // write result for this block to global mem
  if (tid == 0)  {
     g_odata[blockIdx.x] = sdata[0];
  }
}


// Host code
int main(int argc, char** argv)
{
    ParseArguments(argc, argv);

    int N = GlobalSize;
    printf("Vector reduction: size %d\n", N);
    size_t in_size = N * sizeof(float);
    float CPU_result = 0.0, GPU_result = 0.0;

    // Allocate input vectors h_In and h_B in host memory
    h_In = (float*)malloc(in_size);
    if (h_In == 0) 
      Cleanup();

    // Initialize input vectors
    RandomInit(h_In, N);

    //create timer
    cutilCheckError(cutCreateTimer(&timer_mem));
    cutilCheckError(cutCreateTimer(&timer_total));
    cutilCheckError(cutCreateTimer(&timer_GPU));
    cutilCheckError(cutCreateTimer(&timer_CPU));

    // Set the kernel arguments
    int threadsPerBlock = BlockSize;
    int sharedMemSize = threadsPerBlock * sizeof(float);
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
    size_t out_size = blocksPerGrid * sizeof(float);

    // Allocate host output
    h_Out = (float*)malloc(out_size);
    if (h_Out == 0) 
      Cleanup();

    // STUDENT: CPU computation - time this routine for base comparison
    
    cutilCheckError(cutStartTimer(timer_CPU));

    CPU_result = CPUReduce(h_In, N);

    cutilCheckError(cutStopTimer(timer_CPU));
    printf("CPU Reduction time: %f (ms) \n",cutGetTimerValue(timer_CPU));

    // Allocate vectors in device memory
    cutilSafeCall( hipMalloc((void**)&d_In, in_size) );
    cutilSafeCall( hipMalloc((void**)&d_Out, out_size) );

    // compute memory transfer time - CPU to GPU
    cutilCheckError(cutStartTimer(timer_mem));
    cutilCheckError(cutStartTimer(timer_total));

    // STUDENT: Copy h_In from host memory to device memory
    cutilSafeCall( hipMemcpy(d_In,h_In,in_size,hipMemcpyHostToDevice));   

    cutilCheckError(cutStopTimer(timer_mem));
    printf("CPU to GPU Transfer Time: %f (ms) \n",cutGetTimerValue(timer_mem));
    // compute gpu execution time   
    cutilCheckError(cutStartTimer(timer_GPU));

    // Invoke kernel
    VecReduce<<<blocksPerGrid, threadsPerBlock, sharedMemSize>>>(d_In, d_Out, N);
    cutilCheckMsg("kernel launch failure");
    cutilSafeCall( hipDeviceSynchronize() ); // Have host wait for kernel

    // gpu exec timer stop
    cutilCheckError(cutStopTimer(timer_GPU));

    //compute memmory transger time - GPU to CPU
    cutilCheckError(cutCreateTimer(&timer_mem));
    cutilCheckError(cutStartTimer(timer_mem));

    // STUDENT: copy results back from GPU to the h_Out
    cutilSafeCall( hipMemcpy(h_Out,d_Out,out_size,hipMemcpyDeviceToHost));

    cutilCheckError(cutStopTimer(timer_mem));
    cutilCheckError(cutStopTimer(timer_total));

    printf("GPU Execution time: %f (ms) \n",cutGetTimerValue(timer_GPU));
    printf("GPU to CPU Transfer Time: %f (ms) \n",cutGetTimerValue(timer_mem));
    printf("Overall Execution Time (Memory + GPU): %f (ms) \n", cutGetTimerValue(timer_total));

    // STUDENT: Perform the CPU addition of partial results
    // update variable GPU_result

    int i;
    cutilCheckError(cutStartTimer(timer_CPU));
   
    for( i= 0;i < blocksPerGrid; i++ ) 
    {
        GPU_result = GPU_result + h_Out[i];
    }  

    cutilCheckError(cutStopTimer(timer_CPU));
    printf("CPU Partial Sums Execution time: %f (ms) \n",cutGetTimerValue(timer_CPU));

    // STUDENT Check results to make sure they are the same
    printf("CPU results : %f\n", CPU_result);
    printf("GPU results : %f\n", GPU_result);
 
    Cleanup();
}

void Cleanup(void)
{
    // Free device memory
    if (d_In)
        hipFree(d_In);
    if (d_Out)
        hipFree(d_Out);

    // Free host memory
    if (h_In)
        free(h_In);
    if (h_Out)
        free(h_Out);
    
    cutilCheckError(cutDeleteTimer(timer_GPU));        
    cutilCheckError(cutDeleteTimer(timer_CPU));        
    cutilCheckError(cutDeleteTimer(timer_mem));        
    cutilCheckError(cutDeleteTimer(timer_total));
        
    cutilSafeCall( hipDeviceReset() );
    
    exit(0);
}

// Allocates an array with random float entries.
void RandomInit(float* data, int n)
{
    for (int i = 0; i < n; i++)
        data[i] = rand() / (float)RAND_MAX;
}

void PrintArray(float* data, int n)
{
    for (int i = 0; i < n; i++)
        printf("[%d] => %f\n",i,data[i]);
}

float CPUReduce(float* data, int n)
{
  float sum = 0;
    for (int i = 0; i < n; i++)
        sum = sum + data[i];

  return sum;
}

// Parse program arguments
void ParseArguments(int argc, char** argv)
{
    for (int i = 0; i < argc; ++i) {
        if (strcmp(argv[i], "--size") == 0 || strcmp(argv[i], "-size") == 0) {
                  GlobalSize = atoi(argv[i+1]);
		  i = i + 1;
        }
        if (strcmp(argv[i], "--blocksize") == 0 || strcmp(argv[i], "-blocksize") == 0) {
                  BlockSize = atoi(argv[i+1]);
		  i = i + 1;
	}
    }
}
