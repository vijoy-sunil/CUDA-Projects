#include "hip/hip_runtime.h"

#ifndef _FILTER_KERNEL_H_
#define _FILTER_KERNEL_H_

__global__ void SobelFilter(unsigned char* g_DataIn, unsigned char* g_DataOut, int width, int height)
{
   __shared__ unsigned char sharedMem[BLOCK_HEIGHT * BLOCK_WIDTH];
   float s_SobelMatrix[9];

    s_SobelMatrix[0] = -1;
    s_SobelMatrix[1] = 0;
    s_SobelMatrix[2] = 1;

    s_SobelMatrix[3] = -2;
    s_SobelMatrix[4] = 0;
    s_SobelMatrix[5] = 2;

    s_SobelMatrix[6] = -1;
    s_SobelMatrix[7] = 0;
    s_SobelMatrix[8] = 1;

   // Computer the X and Y global coordinates
   int x = blockIdx.x * TILE_WIDTH + threadIdx.x ;//- FILTER_RADIUS;
   int y = blockIdx.y * TILE_HEIGHT + threadIdx.y ;//- FILTER_RADIUS;

   // Get the Global index into the original image
   int index = y * (width) + x;

   // Perform the first load of values into shared memory
   int sharedIndex = threadIdx.y * blockDim.y + threadIdx.x;
   sharedMem[sharedIndex] = g_DataIn[index];
   __syncthreads();

   // STUDENT:  Check 1
   // Handle the extra thread case where the image width or height 
   // 

   if (x >= width || y >= height)
      return;

   // STUDENT: Check 2
   // Handle the border cases of the global image
   if( x < FILTER_RADIUS || y < FILTER_RADIUS) {
       g_DataOut[index] = g_DataIn[index];
       return;
    }

   if ((x > width - FILTER_RADIUS - 1)&&(x <width)) {
       g_DataOut[index] = g_DataIn[index];
       return;
    }

    if ((y > height - FILTER_RADIUS - 1)&&(y < height)) {
       g_DataOut[index] = g_DataIn[index];
       return;
    }

    if(x == blockIdx.x * TILE_WIDTH) //left
       return;

    if(x > blockIdx.x * TILE_WIDTH + TILE_WIDTH) //right
       return;

    if(y == blockIdx.y * TILE_HEIGHT) //top
       return;

    if(y > blockIdx.y * TILE_HEIGHT + TILE_HEIGHT)
       return; //bottom
   
   // STUDENT: Make sure only the thread ids should write the sum of the neighbors.
                // g_DataOut[index] = abs(sumX) + abs(sumY) > EDGE_VALUE_THRESHOLD ? 255 : 0;
    float sumX = 0,sumY = 0;
    for(int dy = -FILTER_RADIUS; dy <= FILTER_RADIUS; dy++)
    {
        for(int dx = -FILTER_RADIUS; dx <= FILTER_RADIUS; dx++)
        {
            int my_index = sharedIndex + (dy * blockDim.x + dx);
            float Pixel =  (float)(sharedMem[my_index]);
            //int my_index = y * width + x + (dy * width + dx);

            sumX += Pixel * s_SobelMatrix[(dy + FILTER_RADIUS) * FILTER_DIAMETER + (dx + FILTER_RADIUS)];

            sumY += Pixel * s_SobelMatrix[(dx + FILTER_RADIUS) * FILTER_DIAMETER + (dy + FILTER_RADIUS)];
        }
    }
    g_DataOut[index] = abs(sumX) + abs(sumY) > EDGE_VALUE_THRESHOLD ? 255 : 0;

}


__global__ void AverageFilter(unsigned char* g_DataIn, unsigned char* g_DataOut, int width, int height)
{
    __shared__ unsigned char sharedMem[BLOCK_HEIGHT*BLOCK_WIDTH];

   int x = blockIdx.x * TILE_WIDTH + threadIdx.x ;//- FILTER_RADIUS;
   int y = blockIdx.y * TILE_HEIGHT + threadIdx.y ;//- FILTER_RADIUS;

   // Get the Global index into the original image
   int index = y * (width) + x;

  // STUDENT: write code for Average Filter : use Sobel as base code
  // Perform the first load of values into shared memory
   int sharedIndex = threadIdx.y * blockDim.y + threadIdx.x;
   sharedMem[sharedIndex] = g_DataIn[index];
   __syncthreads();

   // STUDENT:  Check 1
   // Handle the extra thread case where the image width or height 
   // 

   if (x >= width || y >= height)
      return;

   // STUDENT: Check 2
   // Handle the border cases of the global image
   if( x < FILTER_RADIUS || y < FILTER_RADIUS) {
       g_DataOut[index] = g_DataIn[index];
       return;
    }

   if ((x > width - FILTER_RADIUS - 1)&&(x <width)) {
       g_DataOut[index] = g_DataIn[index];
       return;
    }

    if ((y > height - FILTER_RADIUS - 1)&&(y < height)) {
       g_DataOut[index] = g_DataIn[index];
       return;
    }

    if(x == blockIdx.x * TILE_WIDTH) //left
       return;

    if(x > blockIdx.x * TILE_WIDTH + TILE_WIDTH) //right
       return;

    if(y == blockIdx.y * TILE_HEIGHT) //top
       return;

    if(y > blockIdx.y * TILE_HEIGHT + TILE_HEIGHT)
       return; //bottom
   
   // STUDENT: Make sure only the thread ids should write the sum of the neighbors.

    float sum = 0;
    for(int dy = -FILTER_RADIUS; dy <= FILTER_RADIUS; dy++)
    {
        for(int dx = -FILTER_RADIUS; dx <= FILTER_RADIUS; dx++)
        {
            int my_index = sharedIndex + (dy * blockDim.x + dx);
            float Pixel =  (float)(sharedMem[my_index]);            

            sum += Pixel; 

        }
    }
    g_DataOut[index] = (unsigned char)(sum/FILTER_AREA);
}



__global__ void HighBoostFilter(unsigned char* g_DataIn, unsigned char* g_DataOut, int width, int height)
{
   __shared__ unsigned char sharedMem[BLOCK_HEIGHT*BLOCK_WIDTH];

   int x = blockIdx.x * TILE_WIDTH + threadIdx.x ;//- FILTER_RADIUS;
   int y = blockIdx.y * TILE_HEIGHT + threadIdx.y ;//- FILTER_RADIUS;

   // Get the Global index into the original image
   int index = y * (width) + x;


   // Perform the first load of values into shared memory
   int sharedIndex = threadIdx.y * blockDim.y + threadIdx.x;
   sharedMem[sharedIndex] = g_DataIn[index];
   __syncthreads();

   // STUDENT:  Check 1
   // Handle the extra thread case where the image width or height 
   // 

   if (x >= width || y >= height)
      return;

   // STUDENT: Check 2
   // Handle the border cases of the global image
   if( x < FILTER_RADIUS || y < FILTER_RADIUS) {
       g_DataOut[index] = g_DataIn[index];
       return;
    }

   if ((x > width - FILTER_RADIUS - 1)&&(x <width)) {
       g_DataOut[index] = g_DataIn[index];
       return;
    }

    if ((y > height - FILTER_RADIUS - 1)&&(y < height)) {
       g_DataOut[index] = g_DataIn[index];
       return;
    }

    if(x == blockIdx.x * TILE_WIDTH) //left
       return;

    if(x > blockIdx.x * TILE_WIDTH + TILE_WIDTH) //right
       return;

    if(y == blockIdx.y * TILE_HEIGHT) //top
       return;

    if(y > blockIdx.y * TILE_HEIGHT + TILE_HEIGHT)
       return; //bottom
   
   // STUDENT: Make sure only the thread ids should write the sum of the neighbors.
  // STUDENT: write code for High Boost Filter : use Sobel as base code

    unsigned char centerPixel = sharedMem[sharedIndex];

    float sum = 0;
    for(int dy = -FILTER_RADIUS; dy <= FILTER_RADIUS; dy++)
    {
        for(int dx = -FILTER_RADIUS; dx <= FILTER_RADIUS; dx++)
        {
            int my_index = sharedIndex + (dy * blockDim.x + dx);
            float Pixel =  (float)(sharedMem[my_index]);            

            sum += Pixel; 

        }
    }
    g_DataOut[index] = CLAMP_8bit(centerPixel + HIGH_BOOST_FACTOR * (unsigned char)(centerPixel - sum/FILTER_AREA));
}


#endif // _FILTER_KERNEL_H_


