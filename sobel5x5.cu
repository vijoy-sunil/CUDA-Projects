#include "hip/hip_runtime.h"
// 
// Filters
//

// Includes: system
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <sys/types.h>
#include <sys/stat.h>
#include <unistd.h>
#include <fcntl.h>
#include <stdint.h>
#include <errno.h>
#include <assert.h>
#include <string.h>
#include <sys/io.h>

#include <cutil_inline.h>

// Includes: local
#include "bmp.h"
float SobelMatrix[25] = {1, 2,  0, -2,  -1,
                         4, 8,  0, -8,  -4,
                         6, 12, 0, -12, -6,
                         4, 8,  0, -8,  -4,
                         1, 2,  0, -2,  -1 };


float *s_SobelMatrix;
enum {SOBEL_FILTER=1, AVERAGE_FILTER, HIGH_BOOST_FILTER};

#define CLAMP_8bit(x) max(0, min(255, (x)))

char *BMPInFile = "/home/students/visu3975/NVIDIA_GPU_Computing_SDK/C/src/Filter/lena.bmp";
char *BMPOutFile = "/home/students/visu3975/NVIDIA_GPU_Computing_SDK/C/src/Filter/lena5x5.bmp";
char *Filter = "sobel";
//char *Filter = "average";
//char *Filter = "boost";

int FilterMode  = SOBEL_FILTER;
//int FilterMode  = AVERAGE_FILTER;
//int FilterMode  = HIGH_BOOST_FILTER;

// Functions
void Cleanup(void);
void ParseArguments(int, char**);
void FilterWrapper(unsigned char* pImageIn, int Width, int Height);

// Kernels
__global__ void SobelFilter(const float* SobelMatrix,unsigned char *g_DataIn, unsigned char *g_DataOut, int width, int height);
//__global__ void AverageFilter(unsigned char *g_DataIn, unsigned char *g_DataOut, int width, int height);
//__global__ void HighBoostFilter(unsigned char *g_DataIn, unsigned char *g_DataOut, int width, int height);

/* Device Memory */
unsigned char *d_In;
unsigned char *d_Out;

// Setup for kernel size
const int TILE_WIDTH    = 4;
const int TILE_HEIGHT   = 4;

const int FILTER_RADIUS = 2;
//  const int FILTER_RADIUS = 3;

const int FILTER_DIAMETER = 2 * FILTER_RADIUS + 1;
//const int FILTER_AREA   = FILTER_DIAMETER * FILTER_DIAMETER;

const int BLOCK_WIDTH   = TILE_WIDTH + 2*FILTER_RADIUS;
const int BLOCK_HEIGHT  = TILE_HEIGHT + 2*FILTER_RADIUS;

const int EDGE_VALUE_THRESHOLD = 70;
//const int HIGH_BOOST_FACTOR = 10;

#include "filter_kernel5x5.cu"

//timers
unsigned int timer_mem = 0;
unsigned int timer_CPU = 0;
unsigned int timer_GPU = 0;

void BitMapRead(char *file, struct bmp_header *bmp, struct dib_header *dib, unsigned char **data, unsigned char **palete)
{
   size_t palete_size;
   int fd;

   if((fd = open(file, O_RDONLY )) < 0)
           FATAL("Open Source");

   if(read(fd, bmp, BMP_SIZE) != BMP_SIZE)
           FATAL("Read BMP Header");

   if(read(fd, dib, DIB_SIZE) != DIB_SIZE)
           FATAL("Read DIB Header");

   assert(dib->bpp == 8);

   palete_size = bmp->offset - BMP_SIZE - DIB_SIZE;
   if(palete_size > 0) {
           *palete = (unsigned char *)malloc(palete_size);
           int go = read(fd, *palete, palete_size);
           if (go != palete_size) {
                   FATAL("Read Palete");
           }
   }

   *data = (unsigned char *)malloc(dib->image_size);
   if(read(fd, *data, dib->image_size) != dib->image_size)
           FATAL("Read Image");

   close(fd);
}


void BitMapWrite(char *file, struct bmp_header *bmp, struct dib_header *dib, unsigned char *data, unsigned char *palete)
{
   size_t palete_size;
   int fd;

   palete_size = bmp->offset - BMP_SIZE - DIB_SIZE;

   if((fd = open(file, O_WRONLY | O_CREAT | O_TRUNC,
                             S_IRUSR | S_IWUSR |S_IRGRP)) < 0)
           FATAL("Open Destination");

   if(write(fd, bmp, BMP_SIZE) != BMP_SIZE)
           FATAL("Write BMP Header");

   if(write(fd, dib, DIB_SIZE) != DIB_SIZE)
           FATAL("Write BMP Header");

   if(palete_size != 0) {
           if(write(fd, palete, palete_size) != palete_size)
                   FATAL("Write Palete");
   }
   if(write(fd, data, dib->image_size) != dib->image_size)
           FATAL("Write Image");
   close(fd);
}



void CPU_Sobel(unsigned char* imageIn, unsigned char* imageOut, int width, int height)
{
  int i, j, rows, cols, startCol, endCol, startRow, endRow;
  rows = height;
  cols = width;
 
  // Initialize all output pixels to zero 
  for(i=0; i<rows; i++) {
    for(j=0; j<cols; j++) {
	imageOut[i*width + j] = 0;
    }
  }

  startCol = 2;
  endCol = cols - 2;
  startRow = 2;
  endRow = rows - 2;
  
  // Go through all inner pizel positions 
  for(i=startRow; i<endRow; i++) {
    for(j=startCol; j<endCol; j++) {

       // sum up the 9 values to calculate both the direction x and direction y
       float sumX = 0, sumY=0;
       for(int dy = -FILTER_RADIUS; dy <= FILTER_RADIUS; dy++) {
          for(int dx = -FILTER_RADIUS; dx <= FILTER_RADIUS; dx++) {
             float Pixel = (float)(imageIn[i*width + j +  (dy * width + dx)]);
             sumX += Pixel * SobelMatrix[(dy + FILTER_RADIUS) * FILTER_DIAMETER + (dx+FILTER_RADIUS)];
             sumY += Pixel * SobelMatrix[(dx + FILTER_RADIUS) * FILTER_DIAMETER + (dy+FILTER_RADIUS)];
          }
	}
       imageOut[i*width + j] = (abs(sumX) + abs(sumY)) > EDGE_VALUE_THRESHOLD ? 255 : 0;
    }
  }
}




// Host code
int main(int argc, char** argv)
{
    ParseArguments(argc, argv);

    struct bmp_header bmp;
    struct dib_header dib;

    unsigned char *palete = NULL;
    unsigned char *data = NULL, *out = NULL;

    cutilCheckError(cutCreateTimer(&timer_mem));
    cutilCheckError(cutCreateTimer(&timer_CPU));
    cutilCheckError(cutCreateTimer(&timer_GPU));

    printf("Running %s filter\n", Filter);
    BitMapRead(BMPInFile, &bmp, &dib, &data, &palete);
    out = (unsigned char *)malloc(dib.image_size);

    printf("Computing the CPU output\n");
    printf("Image details: %d by %d = %d , imagesize = %d\n", dib.width, dib.height, dib.width * dib.height,dib.image_size);


    cutilCheckError(cutStartTimer(timer_CPU));
    CPU_Sobel(data, out, dib.width, dib.height);
    cutilCheckError(cutStopTimer(timer_CPU));

    BitMapWrite("CPU_sobel.bmp", &bmp, &dib, out, palete);
    printf("Done with CPU output\n");

    printf("CPU Execution time (Sobel Filter): %f (ms) \n",cutGetTimerValue(timer_CPU));

    printf("Allocating %d bytes for image \n", dib.image_size);
    cutilSafeCall( hipMalloc( (void **)&d_In, dib.image_size*sizeof(unsigned char)) );
    cutilSafeCall( hipMalloc( (void **)&d_Out, dib.image_size*sizeof(unsigned char)) );

//----------------------------------------------------------------
    cutilSafeCall( hipMalloc( (void **)&s_SobelMatrix, 25*sizeof(float)));
    hipMemcpy(s_SobelMatrix, SobelMatrix, 25*sizeof(float), hipMemcpyHostToDevice);

//----------------------------------------------------------------
    cutilCheckError(cutStartTimer(timer_mem));
    hipMemcpy(d_In, data, dib.image_size*sizeof(unsigned char), hipMemcpyHostToDevice);
    
    cutilCheckError(cutStopTimer(timer_mem));
    printf("CPU to GPU Transfer Time: %f (ms) \n",cutGetTimerValue(timer_mem));
    
    FilterWrapper(data, dib.width, dib.height);


    cutilCheckError(cutCreateTimer(&timer_mem));
    cutilCheckError(cutStartTimer(timer_mem));
    // Copy image back to host
    hipMemcpy(out, d_Out, dib.image_size*sizeof(unsigned char), hipMemcpyDeviceToHost);
    cutilCheckError(cutStopTimer(timer_mem)); 
    printf("GPU to CPU Transfer Time: %f (ms) \n",cutGetTimerValue(timer_mem));
    // Write output image   
    BitMapWrite(BMPOutFile, &bmp, &dib, out, palete);

    Cleanup();
}

void Cleanup(void)
{
    cutilSafeCall( hipDeviceReset() );
    exit(0);
}


void FilterWrapper(unsigned char* pImageIn, int Width, int Height)
{
   // Design grid disection around tile size
   int gridWidth  = (Width + TILE_WIDTH - 1) / TILE_WIDTH;
   int gridHeight = (Height + TILE_HEIGHT - 1) / TILE_HEIGHT;
   dim3 dimGrid(gridWidth, gridHeight);

   // But actually invoke larger blocks to take care of surrounding shared memory
   dim3 dimBlock(BLOCK_WIDTH, BLOCK_HEIGHT);

//   switch(FilterMode) {
//     case SOBEL_FILTER:
     printf("Sobel Filter \n");

     cutilCheckError(cutStartTimer(timer_GPU)); 
     SobelFilter<<< dimGrid, dimBlock >>>(s_SobelMatrix,d_In, d_Out, Width, Height);     
     cutilCheckError(cutStopTimer(timer_GPU));
     printf("SOBEL GPU Execution Time: %f (ms) \n",cutGetTimerValue(timer_GPU));
     cutilCheckMsg("kernel launch failure");
//     break;
//     case AVERAGE_FILTER:
//     printf("Average Filter \n");
//     AverageFilter<<< dimGrid, dimBlock >>>(d_In, d_Out, Width, Height);
//     cutilCheckMsg("kernel launch failure");
//     break;
//     case HIGH_BOOST_FILTER:
//     printf("Boost Filter \n");
//     HighBoostFilter<<< dimGrid, dimBlock >>>(d_In, d_Out, Width, Height);
//     cutilCheckMsg("kernel launch failure");
//     break;
//    }
   cutilSafeCall( hipDeviceSynchronize() );
}



// Parse program arguments
void ParseArguments(int argc, char** argv)
{
    for (int i = 0; i < argc; ++i) {
        if (strcmp(argv[i], "--file") == 0 || strcmp(argv[i], "-file") == 0) {
            BMPInFile = argv[i+1];
	    i = i + 1;
        }
        if (strcmp(argv[i], "--out") == 0 || strcmp(argv[i], "-out") == 0) {
            BMPOutFile = argv[i+1];
	    i = i + 1;
        }
        if (strcmp(argv[i], "--filter") == 0 || strcmp(argv[i], "-filter") == 0) {
            Filter = argv[i+1];
	    i = i + 1;
            if (strcmp(Filter, "sobel") == 0)
		FilterMode = SOBEL_FILTER;
            else if (strcmp(Filter, "average") == 0)
		FilterMode = AVERAGE_FILTER;
            else if (strcmp(Filter, "boost") == 0)
		FilterMode = HIGH_BOOST_FILTER;
	 
        }
    }
}



